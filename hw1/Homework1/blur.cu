#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>
#include <atlimage.h>

const double PI = 3.14159265358979323846;

const int BLOCKDIM = 32;

// Note: this must be an odd number
__device__ const int FILTER_SIZE = 15;
__device__ const int FILTER_RADIUS = FILTER_SIZE >> 1;

__device__ const int BLUE_MASK = 0x00ff0000;
__device__ const int GREEN_MASK = 0x0000ff00;
__device__ const int RED_MASK = 0x000000ff;

/** compute index into an int* for the pixel (x,y) in the given 2D pitched allocation */
__device__ int index(int x, int y, const hipPitchedPtr& cpp) {
	// divide by 4 because each pixel is 4B and cpp.pitch is in bytes, but we need to return an index
	return (y * (cpp.pitch / 4)) + x;
}

/** Clamp the given value to the interval [0,bound) */
__device__ int clamp(int value, int bound) {
	if (value < 0) {
		return 0;
	}
	if (value < bound) {
		return value;
	}
	return bound - 1;
}

/** Compute a Gaussian blur of src image and place into dst. Use only global memory. */
__global__ void blurGlobal(hipPitchedPtr src, hipPitchedPtr dst, float* gaussian) {

	int x = (blockDim.x * blockIdx.x) + threadIdx.y;
	int y = (blockDim.y * blockIdx.y) + threadIdx.x;

	float r = 0.0, g = 0.0, b = 0.0;

	for (int ky = 0; ky < FILTER_SIZE; ky++) {
		for (int kx = 0; kx < FILTER_SIZE; kx++) {
			// this replicates border pixels
			// why src.xsize /4? is it because xsize is in bytpes and each elem is 32 bits?
			int i = index(clamp(x + kx - FILTER_RADIUS, src.xsize / 4),
				clamp(y + ky - FILTER_RADIUS, src.ysize), src);
			unsigned int pixel = ((int*)src.ptr)[i];
			// convolute each channel separately
			const float k = gaussian[(ky * FILTER_SIZE) + kx];
			b += (float)((pixel & BLUE_MASK) >> 16) * k;
			g += (float)((pixel & GREEN_MASK) >> 8) * k;
			r += (float)((pixel & RED_MASK)) * k;
		}
	}
	// Re-assemble destination pixel
	unsigned int dpixel = 0x00000000
		| ((((int)b) << 16) & BLUE_MASK)
		| ((((int)g) << 8) & GREEN_MASK)
		| (((int)r) & RED_MASK);
	((int*)dst.ptr)[index(x, y, dst)] = dpixel;
}


/** Compute a Gaussian blur of src image and place into dst. Use only global memory. */
__global__ void blurGlobalPart1(hipPitchedPtr src, hipPitchedPtr dst, float* gaussian) {

	int x = (blockDim.x * blockIdx.x) + threadIdx.y;
	int y = (blockDim.y * blockIdx.y) + threadIdx.x;

	float r = 0.0, g = 0.0, b = 0.0;

	for (int ky = 0; ky < FILTER_SIZE; ky++) {
		for (int kx = 0; kx < FILTER_SIZE; kx++) {
			// this replicates border pixels
			// why src.xsize /4? is it because xsize is in bytpes and each elem is 32 bits?
			int i = index(clamp(x + kx - FILTER_RADIUS, src.xsize / 4),
				clamp(y + ky - FILTER_RADIUS, src.ysize), src);
			unsigned int pixel = ((int*)src.ptr)[i];
			// convolute each channel separately
			const float k = gaussian[(ky * FILTER_SIZE) + kx];
			b += (float)((pixel & BLUE_MASK) >> 16) * k;
			g += (float)((pixel & GREEN_MASK) >> 8) * k;
			r += (float)((pixel & RED_MASK)) * k;
		}
	}
	// Re-assemble destination pixel
	unsigned int dpixel = 0x00000000
		| ((((int)b) << 16) & BLUE_MASK)
		| ((((int)g) << 8) & GREEN_MASK)
		| (((int)r) & RED_MASK);
	((int*)dst.ptr)[index(x, y, dst)] = dpixel;
}


/** Compute a Gaussian blur of src image and place into dst. Use only global memory. */
__global__ void blurGlobalPart2(hipPitchedPtr src, hipPitchedPtr dst, float* gaussian) {

	int x = (blockDim.x * blockIdx.x) + threadIdx.y;
	int y = (blockDim.y * blockIdx.y) + threadIdx.x;

	float r = 0.0, g = 0.0, b = 0.0;

	for (int ky = 0; ky < FILTER_SIZE; ky++) {
		for (int kx = 0; kx < FILTER_SIZE; kx++) {
			// this replicates border pixels
			// why src.xsize /4? is it because xsize is in bytpes and each elem is 32 bits?
			int i = index(clamp(x + kx - FILTER_RADIUS, src.xsize / 4),
				clamp(y + ky - FILTER_RADIUS, src.ysize), src);
			unsigned int pixel = ((int*)src.ptr)[i];
			// convolute each channel separately
			const float k = gaussian[(ky * FILTER_SIZE) + kx];
			b += (float)((pixel & BLUE_MASK) >> 16) * k;
			g += (float)((pixel & GREEN_MASK) >> 8) * k;
			r += (float)((pixel & RED_MASK)) * k;
		}
	}
	// Re-assemble destination pixel
	unsigned int dpixel = 0x00000000
		| ((((int)b) << 16) & BLUE_MASK)
		| ((((int)g) << 8) & GREEN_MASK)
		| (((int)r) & RED_MASK);
	((int*)dst.ptr)[index(x, y, dst)] = dpixel;
}


/** Compute a Gaussian blur of src image and place into dst. Use only global memory. */
__global__ void blurGlobalPart3(hipPitchedPtr src, hipPitchedPtr dst, float* gaussian) {

	int x = (blockDim.x * blockIdx.x) + threadIdx.y;
	int y = (blockDim.y * blockIdx.y) + threadIdx.x;

	float r = 0.0, g = 0.0, b = 0.0;

	for (int ky = 0; ky < FILTER_SIZE; ky++) {
		for (int kx = 0; kx < FILTER_SIZE; kx++) {
			// this replicates border pixels
			// why src.xsize /4? is it because xsize is in bytpes and each elem is 32 bits?
			int i = index(clamp(x + kx - FILTER_RADIUS, src.xsize / 4),
				clamp(y + ky - FILTER_RADIUS, src.ysize), src);
			unsigned int pixel = ((int*)src.ptr)[i];
			// convolute each channel separately
			const float k = gaussian[(ky * FILTER_SIZE) + kx];
			b += (float)((pixel & BLUE_MASK) >> 16) * k;
			g += (float)((pixel & GREEN_MASK) >> 8) * k;
			r += (float)((pixel & RED_MASK)) * k;
		}
	}
	// Re-assemble destination pixel
	unsigned int dpixel = 0x00000000
		| ((((int)b) << 16) & BLUE_MASK)
		| ((((int)g) << 8) & GREEN_MASK)
		| (((int)r) & RED_MASK);
	((int*)dst.ptr)[index(x, y, dst)] = dpixel;
}



void setupGaussian(float** d_gaussian) {
	// calculate gaussian blur filter
	float gaussian[FILTER_SIZE][FILTER_SIZE];
	double sigma = 5.0;
	double mean = FILTER_SIZE / 2;
	for (int x = 0; x < FILTER_SIZE; ++x) {
		for (int y = 0; y < FILTER_SIZE; ++y) {
			double g = exp(-0.5 * (pow((x - mean) / sigma, 2.0) + pow((y - mean) / sigma, 2.0))) / (2 * PI * sigma * sigma);
			gaussian[y][x] = (float)g;
		}
	}
	// normalize the filter
	float sum = 0.0;
	for (int x = 0; x < FILTER_SIZE; ++x) {
		for (int y = 0; y < FILTER_SIZE; ++y) {
			sum += gaussian[y][x];
		}
	}
	for (int x = 0; x < FILTER_SIZE; ++x) {
		for (int y = 0; y < FILTER_SIZE; ++y) {
			gaussian[y][x] /= sum;
		}
	}

	// copy gaussian to device memory
	hipError_t cudaStatus = hipMalloc(d_gaussian, FILTER_SIZE * FILTER_SIZE * sizeof(float));
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMemcpy(*d_gaussian, &gaussian[0], FILTER_SIZE * FILTER_SIZE * sizeof(float), hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);
}

// used by startTimer() and stopTimer()
hipEvent_t start, stop;

void startTimer() {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

/** Return elapsed time (in ms) since startTime() was called */
float stopTimer() {
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	return time;
}

int main(int argc, char* argv[]) {

	char smallInput[] = "C:\\Users\\Administrator\\Source\\Repos\\cis601\\hw1\\steel_wool_small.jpg";
	char largeInput[] = "C:\\Users\\Administrator\\Source\\Repos\\cis601\\hw1\\steel_wool_large.jpg";
	char smallOutput[] = "C:\\Users\\Administrator\\Source\\Repos\\cis601\\hw1\\output_small.jpg";
	char largeOutput[] = "C:\\Users\\Administrator\\Source\\Repos\\cis601\\hw1\\output_large.jpg";
	int kernelSelection = 0;
	char *input;
	char *output;

	//INPUT FIGURE PATH
	if (argc < 3)
	{
		printf("[Usage] program kernel_selection(0/1/2/3) input_size(small/large)");
		exit(1);
	}
	kernelSelection = atoi(argv[1]);
	if (!strcmp(argv[2], "small"))
	{
		input = smallInput;
		output = smallOutput;
	}
	else if (!strcmp(argv[2], "large"))
	{
		input = largeInput;
		output = largeOutput;
	}
	printf("[INFO] kernelSelection=%d input=%s output=%s\n", kernelSelection, input, output);

	// LOAD IMAGE FROM FILE
	CImage img;
	//img.Load("C:\\Users\\Administrator\\Source\\Repos\\cis601\\hw1\\steel_wool_small.jpg");
	img.Load(input);

	// ensure that image dimensions are a multiple of the block size
	if (img.GetHeight() % BLOCKDIM != 0) {
		printf("ERROR: image height (%d) must be a multiple of the block size (%d)\n", img.GetHeight(), BLOCKDIM);
		return 1;
	}
	if (img.GetWidth() % BLOCKDIM != 0) {
		printf("ERROR: image width (%d) must be a multiple of the block size (%d)\n", img.GetWidth(), BLOCKDIM);
		return 1;
	}

	const int IMG_WIDTH_BYTES = img.GetWidth() * 4;

	// COLORREF is a 32-bit int of the form 0x00bbggrr (8 bits for each of the blue, green, and red channels)
	COLORREF* h_buf = new COLORREF[img.GetWidth() * img.GetHeight()];

	// fill up h_buf
	for (int y = 0; y < img.GetHeight(); y++) {
		char* imgPixel = (char*)img.GetPixelAddress(0, y);
		for (int x = 0; x < img.GetWidth(); x++, imgPixel += 3) {
			h_buf[(y * img.GetWidth()) + x] = RGB(imgPixel[2], imgPixel[1], imgPixel[0]);
		}
	}

	hipError_t cudaStatus;

	// use 48KB for shared memory, and 16KB for L1D$
	cudaStatus = hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	checkCudaErrors(cudaStatus);

	// ensure kernel timeout is disabled
	int kernelTimeout;
	cudaStatus = hipDeviceGetAttribute(&kernelTimeout, hipDeviceAttributeKernelExecTimeout, 0/*device*/);
	checkCudaErrors(cudaStatus);
	if (kernelTimeout != 0) {
		printf("WARNING: kernel timeout is enabled!\n", kernelTimeout);
	}

	// COPY IMAGE BUFFERS AND FILTER TO DEVICE
	startTimer();
	hipExtent extent = make_hipExtent(IMG_WIDTH_BYTES, img.GetHeight(), 1);
	hipPitchedPtr d_src, d_dst;
	cudaStatus = hipMalloc3D(&d_src, extent);
	checkCudaErrors(cudaStatus);
	cudaStatus = hipMemcpy2D(d_src.ptr, d_src.pitch,
		h_buf, IMG_WIDTH_BYTES, IMG_WIDTH_BYTES, img.GetHeight(),
		hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMalloc3D(&d_dst, extent);
	checkCudaErrors(cudaStatus);
	cudaStatus = hipMemset2D(d_dst.ptr, d_dst.pitch, 0, IMG_WIDTH_BYTES, img.GetHeight());
	checkCudaErrors(cudaStatus);

	float* d_gaussian;
	setupGaussian(&d_gaussian);

	printf("Copy to device:  %3.1f ms \n", stopTimer());

	// LAUNCH KERNEL

	for (int i = 0; i < 5; i++) {
		startTimer();
		dim3 blocksInGrid(img.GetWidth() / BLOCKDIM, img.GetHeight() / BLOCKDIM);
		dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
		if ( kernelSelection == 0 )
		{ 
			blurGlobal << <blocksInGrid, threadsPerBlock >> >(d_src, d_dst, d_gaussian);
		}
		else if (kernelSelection == 1)
		{
			blurGlobalPart1 << <blocksInGrid, threadsPerBlock >> >(d_src, d_dst, d_gaussian);
		}
		else if (kernelSelection == 2)
		{
			blurGlobalPart2 << <blocksInGrid, threadsPerBlock >> >(d_src, d_dst, d_gaussian);
		}
		else if (kernelSelection == 3)
		{
			blurGlobalPart3 << <blocksInGrid, threadsPerBlock >> >(d_src, d_dst, d_gaussian);
		}


		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		checkCudaErrors(cudaStatus);

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		checkCudaErrors(cudaStatus);

		printf("Kernel time:  %3.1f ms \n", stopTimer());
	}

	// COPY  OUTPUT IMAGE BACK TO HOST
	startTimer();
	cudaStatus = hipMemcpy2D(h_buf, IMG_WIDTH_BYTES,
		d_dst.ptr, d_dst.pitch, IMG_WIDTH_BYTES, d_dst.ysize,
		hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);

	printf("Copy from device:  %3.1f ms \n", stopTimer());

	// WRITE OUT UPDATED IMAGE
	for (int y = 0; y < img.GetHeight(); y++) {
		char* imgPixel = (char*)img.GetPixelAddress(0, y);
		for (int x = 0; x < img.GetWidth(); x++, imgPixel += 3) {
			COLORREF c = h_buf[(y * img.GetWidth()) + x];
			imgPixel[2] = c & 0xFF;
			imgPixel[1] = (c >> 8) & 0XFF;
			imgPixel[0] = (c >> 16) & 0xFF;
		}
	}
	img.Save("C:\\Users\\Administrator\\Source\\Repos\\cis601\\hw1\\out.jpg");

	// CLEANUP

	cudaStatus = hipFree(d_src.ptr);
	checkCudaErrors(cudaStatus);
	cudaStatus = hipFree(d_dst.ptr);
	checkCudaErrors(cudaStatus);
	cudaStatus = hipFree(d_gaussian);
	checkCudaErrors(cudaStatus);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	checkCudaErrors(cudaStatus);

	return 0;
}
